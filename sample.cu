#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


void saveFrames(int n, float *U, float *V, int grid_size, int grid_size_whole) {
    unsigned char *img = (unsigned char *)malloc(grid_size_whole);
    for (int i = 0; i < grid_size_whole; i++) {
        float val = V[i];
        if (val < 0) val = 0;
        if (val > 1) val = 1;
        img[i] = (unsigned char)(val * 255);
    }

    char filename[64];
    sprintf(filename, "frames/frame_%04d.png", n/100);
    stbi_write_png(filename, grid_size, grid_size, 1, img, grid_size);
    free(img);
}

void saveFramesCuda(int step, float *d_V, int grid_size, int grid_size_whole) {
    float *h_frame = (float *)malloc(grid_size_whole * sizeof(float));
    if (!h_frame) {
        fprintf(stderr, "Failed to allocate memory for frame.\n");
        return;
    }

    // Copy data from device to host
    hipMemcpy(h_frame, d_V, grid_size_whole * sizeof(float), hipMemcpyDeviceToHost);

    unsigned char *img = (unsigned char *)malloc(grid_size_whole);
    for (int i = 0; i < grid_size_whole; i++) {
        float val = h_frame[i];
        if (val < 0) val = 0;
        if (val > 1) val = 1;
        img[i] = (unsigned char)(val * 255);
    }

    printf("Printing CUDA frame step %d\n", (step/100));

    char filename[64];
    sprintf(filename, "frames/frame_%04d.png", (step/100));
    stbi_write_png(filename, grid_size, grid_size, 1, img, grid_size);

    free(h_frame);
    free(img);
}

void GrayScottSolver(float *U, float *V, float Du, float Dv, float F, float k, float dt, int steps, int grid_size, int grid_size_whole, bool visualize) {

    float *Unew = (float *)malloc(grid_size_whole * sizeof(float));
    float *Vnew = (float *)malloc(grid_size_whole * sizeof(float));

    float lapU, lapV, zmnozek;


    for (int n = 1; n <= steps; n++) {
        for (int i = 1; i < grid_size - 1; i++) {
            for (int j = 1; j < grid_size - 1; j++) {
                int up    = (i - 1 + grid_size) % grid_size;
                int down  = (i + 1) % grid_size;
                int left  = (j - 1 + grid_size) % grid_size;
                int right = (j + 1) % grid_size;

                int index       = i * grid_size + j;
                int index_up    = up * grid_size + j;
                int index_down  = down * grid_size + j;
                int index_left  = i * grid_size + left;
                int index_right = i * grid_size + right;

                lapU = U[index_left] + U[index_right] + U[index_up] + U[index_down] - 4 * U[index];
                lapV = V[index_left] + V[index_right] + V[index_up] + V[index_down] - 4 * V[index];

                zmnozek = U[index] * V[index] * V[index];

                Unew[index] = U[index] + dt * (-zmnozek + F * (1 - U[index]) + Du * lapU);
                Vnew[index] = V[index] + dt * (zmnozek - (F + k) * V[index] + Dv * lapV);
            }
        }

        // memcpy(U, U_new, grid_size_whole);
        // memcpy(V, V_new, grid_size_whole);

        for (int i = 1; i < grid_size - 1; i++) {
            for (int j = 1; j < grid_size - 1; j++) {
                int index = i * grid_size + j;

                U[index] = Unew[index];
                V[index] = Vnew[index];
            }
        }

        if (visualize && n % 100 == 0) {
            saveFrames(n, U, V, grid_size, grid_size_whole);
        }
    }

    free(Unew);
    free(Vnew);
}

__global__ void GrayScottKernel(float *U, float *V,
                                float *Unew, float *Vnew,
                                float Du, float Dv, float F, float k, float dt,
                                int grid_size) {

    // Define shared memory for the tile
    // Add outer cells (1 extra on each side)
    extern __shared__ float shared_mem[];
    float *s_U = shared_mem;
    float *s_V = &s_U[(blockDim.y + 2) * (blockDim.x + 2)];

    // Calculate the global and local thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;  // Starting row
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;  // Starting column

    int tx = threadIdx.x + 1;  // Local x-coordinate with outer offset
    int ty = threadIdx.y + 1;  // Local y-coordinate with outer offset

    // Local indices for shared memory with outer cells
    int s_width = blockDim.x + 2;
    int s_idx = ty * s_width + tx;

    // Global index
    int g_idx = i * grid_size + j;

    // Load data into shared memory
    // Each thread loads its own cell
    if (i < grid_size - 1 && j < grid_size - 1) {
        s_U[s_idx] = U[g_idx];
        s_V[s_idx] = V[g_idx];

        // Load cells (top, bottom, left, right)
        // Top
        if (threadIdx.y == 0) {
            int up = (i - 1 + grid_size) % grid_size;
            s_U[(ty-1) * s_width + tx] = U[up * grid_size + j];
            s_V[(ty-1) * s_width + tx] = V[up * grid_size + j];
        }

        // Bottom
        if (threadIdx.y == blockDim.y - 1 || i == grid_size - 2) {
            int down = (i + 1) % grid_size;
            s_U[(ty+1) * s_width + tx] = U[down * grid_size + j];
            s_V[(ty+1) * s_width + tx] = V[down * grid_size + j];
        }

        // Left
        if (threadIdx.x == 0) {
            int left = (j - 1 + grid_size) % grid_size;
            s_U[ty * s_width + (tx-1)] = U[i * grid_size + left];
            s_V[ty * s_width + (tx-1)] = V[i * grid_size + left];
        }

        // Right
        if (threadIdx.x == blockDim.x - 1 || j == grid_size - 2) {
            int right = (j + 1) % grid_size;
            s_U[ty * s_width + (tx+1)] = U[i * grid_size + right];
            s_V[ty * s_width + (tx+1)] = V[i * grid_size + right];
        }
    }

    __syncthreads();

    // Only process interior cells (exclude borders)
    if (i < grid_size - 1 && j < grid_size - 1) {

        float lapU = s_U[s_idx - 1] +
                     s_U[s_idx + 1] +
                     s_U[s_idx - s_width] +
                     s_U[s_idx + s_width] -
                     4 * s_U[s_idx];

        float lapV = s_V[s_idx - 1] +
                     s_V[s_idx + 1] +
                     s_V[s_idx - s_width] +
                     s_V[s_idx + s_width] -
                     4 * s_V[s_idx];

        float zmnozek = s_U[s_idx] * s_V[s_idx] * s_V[s_idx];

        Unew[g_idx] = s_U[s_idx] + dt * (-zmnozek + F * (1 - s_U[s_idx]) + Du * lapU);
        Vnew[g_idx] = s_V[s_idx] + dt * (zmnozek - (F + k) * s_V[s_idx] + Dv * lapV);
    }
}

// Function to run the CUDA version and measure execution time
float GrayScottSolverCUDA(float *h_U, float *h_V,
                         float Du, float Dv, float F, float k, float dt,
                         int steps, int grid_size, int grid_size_whole,
                         dim3 block_size, bool visualize) {
    float *d_U, *d_V, *d_Unew, *d_Vnew, *d_temp;
    float elapsed_time = 0.0f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Timing
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_U, grid_size_whole * sizeof(float));
    hipMalloc((void**)&d_V, grid_size_whole * sizeof(float));
    hipMalloc((void**)&d_Unew, grid_size_whole * sizeof(float));
    hipMalloc((void**)&d_Vnew, grid_size_whole * sizeof(float));


    hipMemcpy(d_U, h_U, grid_size_whole * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, grid_size_whole * sizeof(float), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // Also accounting for outer blocks
    dim3 grid_dims((grid_size + block_size.x - 3) / block_size.x,
                  (grid_size + block_size.y - 3) / block_size.y);

    // Shared memory size
    size_t shared_mem_size = 2 * (block_size.x + 2) * (block_size.y + 2) * sizeof(float);

    // Main computation loop
    for (int n = 0; n <= steps; n++) {

        GrayScottKernel<<<grid_dims, block_size, shared_mem_size>>>(
            d_U, d_V, d_Unew, d_Vnew, Du, Dv, F, k, dt, grid_size);

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
            break;
        }

        d_temp = d_U;
        d_U = d_Unew;
        d_Unew = d_temp;

        d_temp = d_V;
        d_V = d_Vnew;
        d_Vnew = d_temp;

        if (visualize && n > 0 && n % 100 == 0) {
           saveFramesCuda(n, d_V, grid_size, grid_size_whole);
        }
    }

    hipMemcpy(h_U, d_U, grid_size_whole, hipMemcpyDeviceToHost);
    hipMemcpy(h_V, d_V, grid_size_whole, hipMemcpyDeviceToHost);

    // Timing
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_Unew);
    hipFree(d_Vnew);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsed_time / 1000.0f;
}

int main(int argc, char *argv[]) {
    // Default parameters
    int grid_size = 0;
    dim3 block_size(16, 16);  // Default block size
    int run_sequential = 0;
    int run_parallel = 0;

    bool visualize = false;  // For saving frames, false when benchmarking

    // Simulation parameters
    float Du = 0.16f;
    float Dv = 0.08f;
    float F  = 0.060f;
    float k  = 0.062f;
    float dt = 1.0f;
    int steps = 5000;

    // Parse command line arguments
    if (argc < 2) {
        printf("USAGE: %s grid_size [-s|-p] [-block x y]\n", argv[0]);
        printf("  grid_size: Size of the simulation grid (NxN)\n");
        printf("  -s: Run sequential version\n");
        printf("  -p: Run parallel CUDA version with benchmarking\n");
        printf("  -block x y: Specify custom block dimensions for CUDA (default: 16x16)\n");
        return 1;
    }

    // Parse grid size (first argument)
    grid_size = atoi(argv[1]);
    if (grid_size <= 0) {
        fprintf(stderr, "Invalid grid size: %d\n", grid_size);
        return 1;
    }

    // Parse remaining flags
    for (int i = 2; i < argc; i++) {
        if (strcmp(argv[i], "-s") == 0) {
            run_sequential = 1;
            run_parallel = 0;
        } else if (strcmp(argv[i], "-p") == 0) {
            run_sequential = 0;
            run_parallel = 1;
        } else if (strcmp(argv[i], "-block") == 0 && i+2 < argc) {
            block_size.x = atoi(argv[i+1]);
            block_size.y = atoi(argv[i+2]);
            i += 2;
        }
    }

    // Default sequential
    if (!run_sequential && !run_parallel) {
        run_sequential = 1;
    }

    int grid_size_whole = grid_size * grid_size;
    printf("Grid size: %d x %d\n", grid_size, grid_size);

    // Run sequential version
    if (run_sequential) {
        // Allocate memory
        float *U = (float *)malloc(grid_size_whole * sizeof(float));
        float *V = (float *)malloc(grid_size_whole * sizeof(float));


        if (!U || !V) {
            fprintf(stderr, "Memory allocation failed.\n");
            return 1;
        }

        // Initialize grid
        memset(U, 1, grid_size_whole);
        memset(V, 0, grid_size_whole);

        for (int i = grid_size / 2 - 10; i < grid_size / 2 + 10; i++) {
            for (int j = grid_size / 2 - 10; j < grid_size / 2 + 10; j++) {
                int index = i * grid_size + j;
                U[index] = 0.75;
                V[index] = 0.25;
            }
        }

        printf("Running sequential version...\n");
        clock_t begin = clock();

        GrayScottSolver(U, V, Du, Dv, F, k, dt, steps, grid_size, grid_size_whole, visualize);

        clock_t end = clock();
        float elapsed_s = ((float)(end - begin) / CLOCKS_PER_SEC);
        printf("Sequential method time: %.3f seconds\n", elapsed_s);

        free(U);
        free(V);
    }

    // Run parallel version with benchmarking
    if (run_parallel) {
        // Define different grid sizes to benchmark
        int grid_sizes[] = {256, 512, 1024, 2048, 4096};
        int num_grid_sizes = 5;

        dim3 block_sizes[] = {
        // Square blocks
        dim3(8, 8),
        dim3(16, 16),
        dim3(32, 32),
        
        // Wide rectangular blocks (width > height)
        dim3(32, 4),
        dim3(32, 8), 
        dim3(32, 16),
        dim3(64, 4), 
        dim3(64, 8), 
        dim3(64, 16),
        dim3(128, 8),
        
        // Tall rectangular blocks (height > width)
        dim3(4, 32),
        dim3(8, 32), 
        dim3(16, 32),
        dim3(4, 64), 
        dim3(8, 64), 
        dim3(16, 64),
        dim3(8, 128),
        
        // "Stripes" - very wide or very tall blocks
        dim3(256, 2),
        dim3(256, 4),
        dim3(2, 256),
        dim3(4, 256),
        
        // Non-power-of-2 configurations
        dim3(24, 24),
        dim3(12, 48),
        dim3(48, 12),
        dim3(18, 36),
        dim3(36, 18),
        dim3(19, 19),
    };
    int num_block_sizes = sizeof(block_sizes) / sizeof(dim3);

        printf("Gray-Scott CUDA Benchmarking\n");
        printf("Running %d steps for each configuration\n\n", steps);

        // Print header
        printf("%-10s %-15s %-15s\n", "Grid Size", "Block Size", "Time (s)");
        printf("----------------------------------------\n");

        // Track the best configuration
        float best_time = INFINITY;
        int best_grid_size = 0;
        dim3 best_block_size(0, 0);

        // If grid size provided, benchmark only that one
        if (grid_size > 0) {
            num_grid_sizes = 1;
            grid_sizes[0] = grid_size;
        }

        // Run benchmarks for each grid size
        for (int i = 0; i < num_grid_sizes; i++) {
            int current_grid_size = grid_sizes[i];
            int current_grid_whole = current_grid_size * current_grid_size;

            printf("\nTesting grid size %dx%d\n", current_grid_size, current_grid_size);
            printf("----------------------------------------\n");

            // Test different block sizes for CUDA version
            for (int j = 0; j < num_block_sizes; j++) {
                dim3 current_block_size = block_sizes[j];

                // Skip invalid configurations (block too large)
                if (current_block_size.x * current_block_size.y > 1024) {
                    printf("%-10d %-3dx%-3d %13s\n", current_grid_size,
                           current_block_size.x, current_block_size.y, "Too large");
                    continue;
                }

                // Allocate memory for CUDA version
                float *U_cuda = (float *)malloc(current_grid_whole * sizeof(float));
                float *V_cuda = (float *)malloc(current_grid_whole * sizeof(float));

                if (!U_cuda || !V_cuda) {
                    fprintf(stderr, "Memory allocation failed for grid size %d\n", current_grid_size);
                    continue;
                }

                // Initialize grid
                memset(U_cuda, 1, current_grid_whole);
                memset(V_cuda, 0, current_grid_whole);

                for (int i = current_grid_size / 2 - 10; i < current_grid_size / 2 + 10; i++) {
                    for (int j = current_grid_size / 2 - 10; j < current_grid_size / 2 + 10; j++) {
                        int index = i * current_grid_size + j;
                        U_cuda[index] = 0.75;
                        V_cuda[index] = 0.25;
                    }
                }

                float cuda_time = GrayScottSolverCUDA(U_cuda, V_cuda, Du, Dv, F, k, dt,
                                                    steps, current_grid_size, current_grid_whole,
                                                    current_block_size, visualize);
                
                printf("%-10d %-3dx %-3d %13.3f\n", current_grid_size, 
                       current_block_size.x, current_block_size.y, cuda_time);
                
                if (cuda_time < best_time) {
                    best_time = cuda_time;
                    best_grid_size = current_grid_size;
                    best_block_size = current_block_size;
                }
                
                free(U_cuda);
                free(V_cuda);
            }
        }
        
        printf("\nOptimal configuration:\n");
        printf("Grid size: %dx%d, Block size: %dx%d, Time: %.3f seconds\n", 
               best_grid_size, best_grid_size, best_block_size.x, best_block_size.y, best_time);
    }
    
    return 0;
}